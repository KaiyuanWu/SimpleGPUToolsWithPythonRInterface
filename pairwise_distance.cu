#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <time.h>
#include "pairwise_distance.h"
const int CUDA_NUM_THREADS = 512;
inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}
 
__global__ void pow2_kernel(const int n, const float* x, float* x2) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x){
	x2[i] = x[i]*x[i];
  }
}

void gpu_gemm(hipblasHandle_t handle, const int nrows, const int ncols, const float* x,
    float* out) {
	float alpha=1.f;
	float beta =0.f;
  	hipblasSsyrk(handle, HIPBLAS_FILL_MODE_UPPER,HIPBLAS_OP_T, nrows, ncols, &alpha, x, ncols,  &beta, out, nrows);
}

void gpu_syr2k(hipblasHandle_t handle, const int nrows, const float* diag, const float* ones, float* dist){
	float alpha = 1.f;
        float beta = -2.f;
	hipblasSsyr2k(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, nrows, 1, &alpha, diag, 1, ones, 1, &beta, dist, nrows);
}

int pairwise_distance_gpu2(float* x, int xnrows, int xncols, float* y, int ynrows, int yncols, float* dist){
        hipblasStatus_t stat;
        hipblasHandle_t handle;
        stat = hipblasCreate(&handle);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
                printf ("CUBLAS initialization failed\n");
                return EXIT_FAILURE;
        }
	float *dev_x, *dev_y, *dev_x2, *dev_y2, *dev_dist, *dev_num_samples,*dev_xnorm, *dev_num_features, *dev_ynorm;
	int max_num_samples = max(xnrows, ynrows);
	int max_num_features = max(xncols, yncols);
	hipError_t cudaStat1 = hipMalloc ((void**)&dev_x, xnrows*xncols*sizeof(float));
	hipError_t cudaStat2 = hipMalloc ((void**)&dev_y, ynrows*yncols*sizeof(float));
	hipError_t cudaStat3 = hipMalloc ((void**)&dev_dist, xnrows*ynrows*sizeof(float));
	hipError_t cudaStat4 = hipMalloc ((void**)&dev_num_samples, max_num_samples*sizeof(float));
	hipError_t cudaStat5 = hipMalloc ((void**)&dev_num_features, max_num_features*sizeof(float));
	hipError_t cudaStat6 = hipMalloc ((void**)&dev_xnorm, xnrows*sizeof(float));
	hipError_t cudaStat7 = hipMalloc ((void**)&dev_ynorm, ynrows*sizeof(float));
	hipError_t cudaStat8 = hipMalloc ((void**)&dev_x2, xnrows*xncols*sizeof(float));
	hipError_t cudaStat9 = hipMalloc ((void**)&dev_y2, ynrows*yncols*sizeof(float));

	if(cudaStat1 != hipSuccess||
  	   cudaStat2 != hipSuccess ||	
	   cudaStat3 != hipSuccess ||	
	   cudaStat4 != hipSuccess ||	
 	   cudaStat5 != hipSuccess ||	
	   cudaStat6 != hipSuccess ||	
	   cudaStat7 != hipSuccess ||
	   cudaStat8 != hipSuccess ||
	   cudaStat9 != hipSuccess){
		printf ("device memory allocation failed x\n"); 
                return EXIT_FAILURE; 
	}
	
	int maxn = max(max_num_samples, max_num_features);
	float* temp = new float[maxn];
	for(int i = 0; i < maxn; i++){
		temp[i] = 1;
	}
	cudaStat1 = hipMemcpy(dev_x, x, xnrows*xncols*sizeof(float), hipMemcpyHostToDevice);
	cudaStat2 = hipMemcpy(dev_y, y, ynrows*yncols*sizeof(float), hipMemcpyHostToDevice);
	cudaStat4 = hipMemcpy(dev_num_samples, temp, max_num_samples*sizeof(float), hipMemcpyHostToDevice);
	cudaStat5 = hipMemcpy(dev_num_features, temp, max_num_features*sizeof(float), hipMemcpyHostToDevice);

	if(cudaStat1 != hipSuccess||                                 
           cudaStat2 != hipSuccess ||
           cudaStat4 != hipSuccess ||
           cudaStat5 != hipSuccess ){
                printf ("device memory allocation failed x\n");
                return EXIT_FAILURE;
        }	
	
	//calculate x.^2
	pow2_kernel<<<GET_BLOCKS(xnrows*xncols), CUDA_NUM_THREADS>>>(xnrows*xncols, dev_x, dev_x2);		
	//calculate y.^2
	pow2_kernel<<<GET_BLOCKS(ynrows*yncols), CUDA_NUM_THREADS>>>(ynrows*yncols, dev_y, dev_y2);		
	//calcuate xnorm
	float alpha = 1.f;
	float beta = 0.f;
	stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, xnrows,  xncols, &alpha, dev_num_features, 1, dev_x2, xncols,  &beta, dev_xnorm, 1);
	if( stat != HIPBLAS_STATUS_SUCCESS){
		printf("fail to xnorm!\n");
		return EXIT_FAILURE;
	}
	//calculate ynorm
	stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, ynrows,  yncols, &alpha, dev_num_features, 1, dev_y2, yncols,  &beta, dev_ynorm, 1);
        if( stat != HIPBLAS_STATUS_SUCCESS){
                printf("fail to ynorm!\n");
                return EXIT_FAILURE;
        }
	//calculate dist = xnorm*I 	
	stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, xnrows, ynrows, 1, &alpha, dev_xnorm, xnrows, dev_num_samples, 1,  &beta, dev_dist, xnrows);
        if( stat != HIPBLAS_STATUS_SUCCESS){
                printf("fail to dist = xnorm*I!\n");
                return EXIT_FAILURE;
        }
	//calculate dist += I*ynorm
	beta = 1.f;
	stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, xnrows, ynrows, 1, &alpha, dev_num_samples, xnrows, dev_ynorm, 1,  &beta, dev_dist, xnrows);
        if( stat != HIPBLAS_STATUS_SUCCESS){
                printf("fail to dist += I*ynorm!\n");
                return EXIT_FAILURE;
        } 
	alpha = -2.f;
	beta = 1.f;
	stat = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, xnrows, ynrows, xncols, &alpha, dev_x, xncols, dev_y, yncols,  &beta, dev_dist, xnrows);
        if( stat != HIPBLAS_STATUS_SUCCESS){
                printf("fail to dist += x*yT!\n");
                return EXIT_FAILURE;
        }
	
	cudaStat1 = hipMemcpy (dist, dev_dist, xnrows*ynrows*sizeof(float),hipMemcpyDeviceToHost);
        if (cudaStat1 != hipSuccess) {
                printf ("device memory copy failed dist \n");
                return EXIT_FAILURE;
        }
	// *dev_x, *dev_y, *dev_x2, *dev_y2, *dev_dist, *dev_num_samples,*dev_xnorm, *dev_num_features, *dev_ynorm
	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_x2);
	hipFree(dev_y2);
	hipFree(dev_dist);
	hipFree(dev_num_samples);
	hipFree(dev_num_features);
	hipFree(dev_xnorm);
	hipFree(dev_ynorm);
	delete[] temp;
        hipblasDestroy(handle);
        return true;
		
}
int pairwise_distance_gpu1(float* x, int nrows, int ncols, float* dist){
    	hipblasStatus_t stat;
	hipblasHandle_t handle;
	stat = hipblasCreate(&handle); 
	if (stat != HIPBLAS_STATUS_SUCCESS) { 
		printf ("CUBLAS initialization failed\n"); 
		return EXIT_FAILURE; 
	}
	float *dev_x, *dev_dist, *dev_ones, *diag, *dev_diag;
	diag = new float[nrows];
	
	hipError_t cudaStat1 = hipMalloc ((void**)&dev_x, nrows*ncols*sizeof(float));
	hipError_t cudaStat2 = hipMalloc ((void**)&dev_ones, nrows*sizeof(float));
	hipError_t cudaStat3 = hipMalloc ((void**)&dev_diag, nrows*sizeof(float));;
	hipError_t cudaStat4 = hipMalloc ((void**)&dev_dist, nrows*nrows*sizeof(float));
	if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess || cudaStat4 != hipSuccess) { 
		printf ("device memory allocation failed x\n"); 
		return EXIT_FAILURE; 
	}


	hipError_t cudaStat = hipMemcpy (dev_x, x, ncols*nrows*sizeof(float), hipMemcpyHostToDevice);
	if (cudaStat != hipSuccess) { 
                printf ("device memory copy failed x\n"); 
                return EXIT_FAILURE; 
        }
	gpu_gemm(handle, nrows, ncols, dev_x, dev_dist);
	
	cudaStat = hipMemcpy (dist, dev_dist, nrows*nrows*sizeof(float),hipMemcpyDeviceToHost);
	if (cudaStat != hipSuccess) {
                printf ("device memory copy failed dist \n");
                return EXIT_FAILURE;
        }
	for(int i = 0; i < nrows; i++)
		diag[i] = dist[i*nrows+i];
	cudaStat = hipMemcpy (dev_diag, diag, nrows*sizeof(float), hipMemcpyHostToDevice);
        if (cudaStat != hipSuccess) { 
                printf ("device memory copy failed diag\n"); 
                return EXIT_FAILURE; 
        }  
	for(int i = 0; i < nrows; i++)
                diag[i] = 1;
	cudaStat = hipMemcpy (dev_ones, diag, nrows*sizeof(float), hipMemcpyHostToDevice);
        if (cudaStat != hipSuccess) {
                printf ("device memory copy failed diag\n");
                return EXIT_FAILURE;
        }
	gpu_syr2k(handle, nrows, dev_diag, dev_ones, dev_dist);	
	
	cudaStat = hipMemcpy (dist, dev_dist, nrows*nrows*sizeof(float),hipMemcpyDeviceToHost);
        if (cudaStat != hipSuccess) {
                printf ("device memory copy failed dist \n");
                return EXIT_FAILURE;
        }
	for(int i = 0; i < nrows; i++){
		for(int j = i+1; j < nrows; j++)
			dist[i*nrows+j] = dist[j*nrows + i];
	}

	hipFree (dev_x);
	hipFree (dev_dist);
	hipFree (dev_diag);
	hipFree (dev_ones);
	delete[] diag;
	hipblasDestroy(handle);
	return true;
}
